#include "hip/hip_runtime.h"
#include "../include/spgsknn.hpp"

#include <iostream>
#include <vector>
#include <chrono>

#include <hip/hip_runtime.h>
#include "hipsparse.h"

#include <omp.h>

using namespace std;

inline void check(hipError_t status, string error) {
  if (status != hipSuccess) {
    cout << error << endl;
    exit(1);
  }
}

inline void check(hipsparseStatus_t status, string error) {
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    cout << error << endl;
    exit(1);
  }
}

__global__
void get_col_norms(int *col_csr, float *val_csr, float *sq_norms, int nnz) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < nnz)
    atomicAdd(&sq_norms[col_csr[i]], val_csr[i] * val_csr[i]);
}

__global__
void add_norms(float *C, float *Q_sq_norms, float *R_sq_norms, int m, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  int i = index / n;
  int j = index % n;

  // Cuda store column major.
  if (i < m && j < n)
    C[i + j * m] = Q_sq_norms[i] - 2.0 * C[i + j * m] + R_sq_norms[j];
}

__global__
void iota_fill(int *indices, int m, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  int i = index / n;
  int j = index % n;

  // Cuda store column major.
  if (i < m && j < n)
    indices[i + j * m] = j;
}

__global__
void bitonic_mergesort_step(float *C, int *indices, int split, int away, int m, int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (index >= n)
    return;
  else if (row >= m)
    return;

  int i_mask = ((1 << 30) - 1) - away;
  int j_mask = away;
  int is_inc_mask = split << 1;

  int i = index & i_mask;
  int j = index | j_mask;

  int is_inc = (index & is_inc_mask) == 0;

  if (index == j)
    return;

  bool need_swap = false;
  need_swap |= (is_inc && (C[row + i * m] > C[row + j * m]));
  need_swap |= (!is_inc && (C[row + i * m] < C[row + j * m]));

  if (need_swap) {
    float tmp_C = C[row + i * m];
    int tmp_indices = indices[row + i * m];

    C[row + i * m] = C[row + j * m];
    C[row + j * m] = tmp_C;

    indices[row + i * m] = indices[row + j * m];
    indices[row + j * m] = tmp_indices;
  }
}

template <class T>
int* bitonic_mergesort(T *C, int m, int n) {
  int *indices;

  check(hipMalloc((void**) &indices, m * n * sizeof(int)),
        "initialize indices");

  iota_fill<<<(m * n + 255) / 256, 256>>>(indices, m, n);

  dim3 blocks(32, 32);
  dim3 grids;
  grids.x = (n + blocks.x - 1) / blocks.x;
  grids.y = (m + blocks.y - 1) / blocks.y;

  for (int split = 1; split < n; split <<= 1)
    for (int away = split; away >= 1; away >>= 1)
      bitonic_mergesort_step<<<grids, blocks>>>(C, indices, split, away, m, n);

  return indices;
}

template <class T>
void k_select(T *distances_device, int *indices_device,
               vector<T> &distances, vector<int> &indices,
               int m, int n, int k) {
  distances.resize(m * k);
  indices.resize(m * k);

  check(hipMemcpy(&distances[0], distances_device, (size_t) ((m * k) * sizeof(T)),
                   hipMemcpyDeviceToHost),
        "copy device to host (distances)");

  check(hipMemcpy(&indices[0], indices_device, (size_t) ((m * k) * sizeof(int)),
                   hipMemcpyDeviceToHost),
        "copy device to host (indices)");
}

template <class T>
void coo_to_csr(const vector<int> &A_row, const vector<int> &A_col, const vector<T> &A_val,
                unsigned int m, hipsparseHandle_t handle,
                int *&row_csr, int *&col_csr, T *&val_csr) {
  int *row_coo = 0;

  check(hipMalloc((void**) &row_coo, A_row.size() * sizeof(T)),
        "coo malloc");

  check(hipMalloc((void**) &row_csr, (m+1) * sizeof(T)),
        "csr row malloc");

  check(hipMalloc((void**) &col_csr, A_row.size() * sizeof(T)),
        "csr col malloc");

  check(hipMalloc((void**) &val_csr, A_row.size() * sizeof(T)),
        "csr val malloc");

  check(hipMemcpy(row_coo, &A_row[0], (size_t) (A_row.size() * sizeof(int)),
                   hipMemcpyHostToDevice),
        "copy to row");

  check(hipMemcpy(col_csr, &A_col[0], (size_t) (A_col.size() * sizeof(int)),
                   hipMemcpyHostToDevice),
        "copy to col");

  check(hipMemcpy(val_csr, &A_val[0],
                   (size_t) (A_col.size() * sizeof(T)), hipMemcpyHostToDevice),
        "copy to val");

  check(hipsparseXcoo2csr(handle, row_coo, A_row.size(),
                         m, row_csr, HIPSPARSE_INDEX_BASE_ZERO),
        "COO to CSR");

  check(hipFree(row_coo), "free coo");
}

template <class T>
T* sparse_to_dense(int *row, int *col, T *val,
                   int m, int n,
                   hipsparseHandle_t handle, hipsparseMatDescr_t desc) {
  T *A;

  check(hipMalloc((void**) &A, m * n * sizeof(T)),
        "dense malloc");

  check(hipsparseScsr2dense(handle, m, n, desc,
                           val, row, col, A, m),
        "CSR to dense");

  return A;
}

template <class T>
T* inner_product(
    int *Q_row_csr, int *Q_col_csr, T *Q_val_csr, unsigned int Q_nnz,
    int *R_row_csr, int *R_col_csr, T *R_val_csr, unsigned int R_nnz,
    unsigned int d, unsigned int m, unsigned int n, hipsparseHandle_t handle) {
  int *C_row_csr = 0;
  int *C_col_csr = 0;
  T *C_val_csr = 0;

  int C_nnz = -1;

  hipsparseMatDescr_t real_sparse_desc = 0;

  check(hipsparseCreateMatDescr(&real_sparse_desc), "create");
  check(hipsparseSetMatType(real_sparse_desc, HIPSPARSE_MATRIX_TYPE_GENERAL), "set 1");
  check(hipsparseSetMatIndexBase(real_sparse_desc, HIPSPARSE_INDEX_BASE_ZERO), "set 2");

  check(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST), "set pointer");

  check(hipMalloc((void**) &C_row_csr, (m+1) * sizeof(int)),
        "malloc row fail");

  check(hipsparseXcsrgemmNnz(handle,
                            HIPSPARSE_OPERATION_TRANSPOSE,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            m, n, d,
                            real_sparse_desc, Q_nnz, Q_row_csr, Q_col_csr,
                            real_sparse_desc, R_nnz, R_row_csr, R_col_csr,
                            real_sparse_desc, C_row_csr, &C_nnz),
        "gemm nnz");

  if (C_nnz == -1)
    exit(1);

  check(hipMalloc((void**) &C_col_csr, C_nnz * sizeof(int)),
        "malloc device col");
  check(hipMalloc((void**) &C_val_csr, C_nnz * sizeof(T)),
        "malloc device val");

  check(hipsparseScsrgemm(handle,
                         HIPSPARSE_OPERATION_TRANSPOSE,
                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                         m, n, d,
                         real_sparse_desc, Q_nnz, Q_val_csr, Q_row_csr, Q_col_csr,
                         real_sparse_desc, R_nnz, R_val_csr, R_row_csr, R_col_csr,
                         real_sparse_desc, C_val_csr, C_row_csr, C_col_csr),
        "gemm");

  T* C = sparse_to_dense(C_row_csr, C_col_csr, C_val_csr,
                         m, n, handle, real_sparse_desc);

  check(hipFree(C_row_csr), "free row csr");
  check(hipFree(C_col_csr), "free col csr");
  check(hipFree(C_val_csr), "free val csr");

  return C;
}

template <class T>
void add_sq_norms(int *Q_col_csr, T *Q_val_csr, int Q_nnz,
                  int *R_col_csr, T *R_val_csr, int R_nnz,
                  T *C, int m, int n, hipsparseHandle_t handle) {
  T *Q_sq_norms = 0;
  T *R_sq_norms = 0;

  check(hipMalloc((void**) &Q_sq_norms, m * sizeof(T)), "coo malloc");
  check(hipMalloc((void**) &R_sq_norms, n * sizeof(T)), "coo malloc");

  get_col_norms<<<(Q_nnz + 255) / 256, 256>>>(Q_col_csr, Q_val_csr, Q_sq_norms, Q_nnz);
  get_col_norms<<<(R_nnz + 255) / 256, 256>>>(R_col_csr, R_val_csr, R_sq_norms, R_nnz);

  add_norms<<<(m * n + 255) / 256, 256>>>(C, Q_sq_norms, R_sq_norms, m, n);
}

template <class T>
void spgsknn(unsigned int d, unsigned int m, unsigned int n, unsigned int k,
             vector<int> &Q_row, vector<int> &Q_col, vector<T> &Q_val,
             vector<int> &R_row, vector<int> &R_col, vector<T> &R_val,
             vector<T> &distances,
             vector<int> &indices) {
  hipsparseHandle_t handle = 0;

  check(hipsparseCreate(&handle), "initialization");

  auto start = chrono::high_resolution_clock::now();

  int *Q_row_csr = 0;
  int *Q_col_csr = 0;
  T *Q_val_csr = 0;

  int *R_row_csr = 0;
  int *R_col_csr = 0;
  T *R_val_csr = 0;

  coo_to_csr(Q_row, Q_col, Q_val, d, handle, Q_row_csr, Q_col_csr, Q_val_csr);
  coo_to_csr(R_row, R_col, R_val, d, handle, R_row_csr, R_col_csr, R_val_csr);

  auto conv_done = chrono::high_resolution_clock::now();

  T *distances_device = inner_product(Q_row_csr, Q_col_csr, Q_val_csr, Q_val.size(),
                                      R_row_csr, R_col_csr, R_val_csr, R_val.size(),
                                      d, m, n, handle);

  auto mult_done = chrono::high_resolution_clock::now();

  add_sq_norms(
      Q_col_csr, Q_val_csr, Q_val.size(),
      R_col_csr, R_val_csr, R_val.size(),
      distances_device, m, n, handle);

  auto norm_done = chrono::high_resolution_clock::now();

  int *indices_device = bitonic_mergesort(distances_device, m, n);

  auto sort_done = chrono::high_resolution_clock::now();

  k_select(distances_device, indices_device, distances, indices, m, n, k);

  auto select_done = chrono::high_resolution_clock::now();

  check(hipFree(distances_device), "free distances");
  check(hipFree(indices_device), "free indices");

  float total = chrono::duration_cast<chrono::milliseconds>(select_done-start).count();
  float conv = chrono::duration_cast<chrono::milliseconds>(conv_done-start).count();
  float mult = chrono::duration_cast<chrono::milliseconds>(mult_done-conv_done).count();
  float norm = chrono::duration_cast<chrono::milliseconds>(norm_done-mult_done).count();
  float sort = chrono::duration_cast<chrono::milliseconds>(sort_done-norm_done).count();
  float select = chrono::duration_cast<chrono::milliseconds>(select_done-sort_done).count();

  cout << "total: " << total / 1000.0 << endl;
  cout << "conv: " << conv / 1000.0 << endl;
  cout << "mult: " << mult / 1000.0 << endl;
  cout << "norm: " << norm / 1000.0 << endl;
  cout << "sort: " << sort / 1000.0 << endl;
  cout << "select: " << select / 1000.0 << endl;
}

// Possible instantiations.
template void spgsknn(unsigned int, unsigned int, unsigned int, unsigned int,
                      vector<int>&, vector<int>&, vector<float>&,
                      vector<int>&, vector<int>&, vector<float>&,
                      vector<float> &, vector<int> &);
