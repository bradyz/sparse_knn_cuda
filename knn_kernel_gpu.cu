#include "hip/hip_runtime.h"
#include "knn_kernel_gpu.h"

#include <algorithm>
#include <cassert>
#include <iostream>
#include <tuple>
#include <vector>
#include <map>
#include <chrono>

#include <thrust/sort.h>
#include <thrust/device_vector.h>

#include <hip/hip_runtime.h>
#include "hipsparse.h"

#include <omp.h>

using namespace std;

template <class T>
using row_col_val = tuple<unsigned int, unsigned int, T>;

inline void check(hipError_t status, string error) {
  if (status != hipSuccess) {
    cout << error << endl;
    exit(1);
  }
}

inline void check(hipsparseStatus_t status, string error) {
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    cout << error << endl;
    exit(1);
  }
}

template <class T>
void sort_by_col_row(vector<row_col_val<T>> &triplets) {
  sort(triplets.begin(), triplets.end(),
      [](const row_col_val<T> &u, const row_col_val<T> &v) {
        if (get<1>(u) == get<1>(v))
          return get<0>(u) < get<0>(v);

        return get<1>(u) < get<1>(v);
      });
}

template <class T>
void populate_sq_norms(const vector<int> &A_col, const vector<T> &A_val,
                       vector<T> &A_sq_norms) {
  for (unsigned int i = 0; i < A_val.size(); i++)
    A_sq_norms[A_col[i]] += A_val[i] * A_val[i];
}

template <class T>
void cpu_sort(const vector<int> &host_row, const vector<int> &host_col,
              const vector<T> &host_val,
              const vector<T> &Q_sq_norms, const vector<T> &R_sq_norms,
              int k) {
  int m = Q_sq_norms.size();
  int n = R_sq_norms.size();

  // Save the inner products.
  map<pair<int, int>, float> values;

  int val_idx = 0;

  for (unsigned int i = 0; i < m; i++) {
    for (int j_idx = host_row[i]; j_idx < host_row[i+1]; j_idx++) {
      int j = host_col[j_idx];

      values[make_pair(i, j)] = host_val[val_idx++];
    }
  }

  // Sort.
  vector<vector<int>> neighbors(m, vector<int>(n));

  #pragma omp parallel for
  for (int i = 0; i < m; i++) {
    iota(neighbors[i].begin(), neighbors[i].end(), 0);

    nth_element(neighbors[i].begin(), neighbors[i].begin() + k, neighbors[i].end(),
        [&](int j_1, int j_2) {
        T dist_1 = Q_sq_norms[i] + -2.0 * values[make_pair(i, j_1)] + R_sq_norms[j_1];
        T dist_2 = Q_sq_norms[i] + -2.0 * values[make_pair(i, j_2)] + R_sq_norms[j_2];

        return dist_1 < dist_2;
    });
  }
}

template <class T>
void gpu_sort(int k) {
/*   int m = Q_sq_norms.size(); */
/*   int n = Q_sq_norms.size(); */
/*  */
/*   int val_idx = 0; */
/*  */
/* #pragma omp parallel for */
/*   for (unsigned int i = 0; i < m; i++) { */
/*     vector<int> idx(n); */
/*     vector<T> dist(n); */
/*  */
    /* cout << omp_get_thread_num() << " " << omp_get_num_threads() << endl; */
    /* cout << i << " " << m << endl; */
/*  */
/*     thrust::device_vector<int> gpu_idx(n); */
/*     thrust::sequence(gpu_idx.begin(), gpu_idx.end()); */
/*  */
/*     thrust::device_vector<T> gpu_dist(dist.begin(), dist.end()); */
/*  */
/*     thrust::sort_by_key(gpu_dist.begin(), gpu_dist.end(), gpu_idx.begin()); */
/*  */
/*     thrust::copy(gpu_idx.begin(), gpu_idx.end(), &idx[0]); */
/*   } */
}

template <class T>
void coo_to_csr(vector<int> &A_row, vector<int> &A_col, vector<T> &A_val,
                unsigned int m, hipsparseHandle_t handle,
                int *&row_csr, int *&col_csr, T *&val_csr) {
  int *row_coo = 0;

  check(hipMalloc((void**) &row_coo, A_row.size() * sizeof(T)),
        "coo malloc failed");

  check(hipMalloc((void**) &row_csr, (m+1) * sizeof(T)),
        "csr row malloc failed");

  check(hipMalloc((void**) &col_csr, A_row.size() * sizeof(T)),
        "csr col malloc failed");

  check(hipMalloc((void**) &val_csr, A_row.size() * sizeof(T)),
        "csr val malloc failed");

  check(hipMemcpy(row_coo, &A_row[0], (size_t) (A_row.size() * sizeof(int)),
                   hipMemcpyHostToDevice),
        "copy to row failed");

  check(hipMemcpy(col_csr, &A_col[0], (size_t) (A_col.size() * sizeof(int)),
                   hipMemcpyHostToDevice),
        "copy to col failed");

  check(hipMemcpy(val_csr, &A_val[0],
                   (size_t) (A_col.size() * sizeof(T)), hipMemcpyHostToDevice),
        "copy to val failed");

  check(hipsparseXcoo2csr(handle, row_coo, A_row.size(),
                         m, row_csr, HIPSPARSE_INDEX_BASE_ZERO),
        "convert failed");

  check(hipFree(row_coo), "free coo");
}

template <class T>
T* sparse_to_dense(int *row, int *col, T *val,
                   int m, int n,
                   hipsparseHandle_t handle, hipsparseMatDescr_t desc) {
  T *A;

  check(hipMalloc((void**) &A, m * n * sizeof(T)),
        "dense malloc failed");

  check(hipsparseScsr2dense(handle, m, n, desc,
                           val, row, col, A, m),
        "csr 2 dense.");

  return A;
}

template <class T>
T* inner_product(
    int *Q_row_csr, int *Q_col_csr, T *Q_val_csr, unsigned int Q_nnz,
    int *R_row_csr, int *R_col_csr, T *R_val_csr, unsigned int R_nnz,
    unsigned int d, unsigned int m, unsigned int n, hipsparseHandle_t handle) {
  int *C_row_csr = 0;
  int *C_col_csr = 0;
  T *C_val_csr = 0;

  int C_nnz = -1;

  hipsparseMatDescr_t real_sparse_desc = 0;

  check(hipsparseCreateMatDescr(&real_sparse_desc), "create failed");
  check(hipsparseSetMatType(real_sparse_desc, HIPSPARSE_MATRIX_TYPE_GENERAL), "set 1 failed");
  check(hipsparseSetMatIndexBase(real_sparse_desc, HIPSPARSE_INDEX_BASE_ZERO), "set 2 failed");

  check(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST), "set pointer");

  check(hipMalloc((void**) &C_row_csr, (m+1) * sizeof(int)),
        "malloc row fail");

  check(hipsparseXcsrgemmNnz(handle,
                            HIPSPARSE_OPERATION_TRANSPOSE,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            m, n, d,
                            real_sparse_desc, Q_nnz, Q_row_csr, Q_col_csr,
                            real_sparse_desc, R_nnz, R_row_csr, R_col_csr,
                            real_sparse_desc, C_row_csr, &C_nnz),
        "gemm nnz");

  if (C_nnz == -1)
    exit(1);

  check(hipMalloc((void**) &C_col_csr, C_nnz * sizeof(int)),
        "malloc device col");
  check(hipMalloc((void**) &C_val_csr, C_nnz * sizeof(T)),
        "malloc device val");

  check(hipsparseScsrgemm(handle,
                         HIPSPARSE_OPERATION_TRANSPOSE,
                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                         m, n, d,
                         real_sparse_desc, Q_nnz, Q_val_csr, Q_row_csr, Q_col_csr,
                         real_sparse_desc, R_nnz, R_val_csr, R_row_csr, R_col_csr,
                         real_sparse_desc, C_val_csr, C_row_csr, C_col_csr),
        "gemm");

  T* C = sparse_to_dense(C_row_csr, C_col_csr, C_val_csr,
                         m, n, handle, real_sparse_desc);

  check(hipFree(C_row_csr), "free row csr");
  check(hipFree(C_col_csr), "free col csr");
  check(hipFree(C_val_csr), "free val csr");

  return C;
}

__global__
void get_col_norms(int *col_csr, float *val_csr, float *sq_norms, int nnz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nnz)
      atomicAdd(&sq_norms[col_csr[i]], val_csr[i] * val_csr[i]);
}

__global__
void add_norms(float *C, float *Q_sq_norms, float *R_sq_norms, int m, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int i = index / n;
    int j = index % n;

    // Cuda store column major.
    if (i < m && j < n)
      C[i + j * m] = Q_sq_norms[i] - 2.0 * C[i + j * m] + R_sq_norms[j];
}

template <class T>
void add_sq_norms(int *Q_col_csr, T *Q_val_csr, int Q_nnz,
                  int *R_col_csr, T *R_val_csr, int R_nnz,
                  T *C, int m, int n, hipsparseHandle_t handle) {
  T *Q_sq_norms = 0;
  T *R_sq_norms = 0;

  check(hipMalloc((void**) &Q_sq_norms, m * sizeof(T)), "coo malloc failed");
  check(hipMalloc((void**) &R_sq_norms, n * sizeof(T)), "coo malloc failed");

  get_col_norms<<<(Q_nnz + 255) / 256, 256>>>(Q_col_csr, Q_val_csr, Q_sq_norms, Q_nnz);
  get_col_norms<<<(R_nnz + 255) / 256, 256>>>(R_col_csr, R_val_csr, R_sq_norms, R_nnz);

  add_norms<<<(m * n + 255) / 256, 256>>>(C, Q_sq_norms, R_sq_norms, m, n);
}

template <class T>
void knn(vector<int> &Q_row, vector<int> &Q_col, vector<T> &Q_val,
         vector<int> &R_row, vector<int> &R_col, vector<T> &R_val,
         unsigned int d, unsigned int m, unsigned int n, unsigned int k) {
  hipsparseHandle_t handle = 0;

  check(hipsparseCreate(&handle), "initialization failed");

  auto start = chrono::high_resolution_clock::now();

  int *Q_row_csr = 0;
  int *Q_col_csr = 0;
  T *Q_val_csr = 0;

  int *R_row_csr = 0;
  int *R_col_csr = 0;
  T *R_val_csr = 0;

  coo_to_csr(Q_row, Q_col, Q_val, d, handle, Q_row_csr, Q_col_csr, Q_val_csr);
  coo_to_csr(R_row, R_col, R_val, d, handle, R_row_csr, R_col_csr, R_val_csr);

  auto conversion_done = chrono::high_resolution_clock::now();

  T *C = inner_product(Q_row_csr, Q_col_csr, Q_val_csr, Q_val.size(),
                       R_row_csr, R_col_csr, R_val_csr, R_val.size(),
                       d, m, n, handle);

  auto mult_done = chrono::high_resolution_clock::now();

  add_sq_norms(
      Q_col_csr, Q_val_csr, Q_val.size(),
      R_col_csr, R_val_csr, R_val.size(),
      C, m, n, handle);

  auto norm_done = chrono::high_resolution_clock::now();

  /* gpu_sort(host_row, host_col, host_val, Q_sq_norms, R_sq_norms, k); */

  auto sort_done = chrono::high_resolution_clock::now();

  vector<float> C_host(m * n);

  check(hipMemcpy(&C_host[0], C, (size_t) ((m * n) * sizeof(T)),
                   hipMemcpyDeviceToHost),
        "copy from failed val asdfasdf");

  /* cout << m << " " << n << endl; */
  /* for (int i = 0; i < m; i++) { */
  /*   for (int j = 0; j < n; j++) { */
  /*     cout << C_host[i + j * m] << " "; */
  /*   } */
  /*   cout << endl; */
  /* } */

  float total = chrono::duration_cast<chrono::milliseconds>(sort_done-start).count();
  float mult = chrono::duration_cast<chrono::milliseconds>(mult_done-start).count();
  float norm = chrono::duration_cast<chrono::milliseconds>(norm_done-mult_done).count();
  float sort = chrono::duration_cast<chrono::milliseconds>(sort_done-norm_done).count();

  cout << "total: " << total / 1000.0 << endl;
  cout << "mult: " << mult / 1000.0 << endl;
  cout << "norm: " << norm / 1000.0 << endl;
  cout << "sort: " << sort / 1000.0 << endl;
}

// Possible instantiations.
template void knn(vector<int>&, vector<int>&, vector<float>&,
                  vector<int>&, vector<int>&, vector<float>&,
                  unsigned int, unsigned int, unsigned int, unsigned int);
